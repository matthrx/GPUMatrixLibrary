#include "hip/hip_runtime.h"
// Must be done... 
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h> 
#include <string>
#include <sstream>
#include <cstdlib>
#include <limits>

// #include "../../GPUOperations.h"
// #include "../initialisation/initialisation.cuh"
#include "statisticOperationsKernel.cuh"
#include "../../GpuMatrix.h"
#include "../generalInformation/generalInformation.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define assertm(exp, msg) assert(((void)msg, exp))
#define THREADS_PER_BLOCK_DIM 16
#define carre(x) (x*x)
#define max(a,b) (((a) > (b)) ? (a) : (b)) 
#define min(a, b) (((a) < (b)) ? (a) : (b))

// struct typeProps deviceProps;
// for (unsigned int i = 0; i < 3; i++){
//      deviceProps.(maxGridSize[i]) = 65355;
// }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
      std::cerr << hipGetErrorString(code) << " file : " << file << " line : " <<  line << std::endl;
      if (abort) { exit(code); }
    }
}

template <typename T> 
T GpuMatrix<T>::minGpuMatrix(void){
    const size_t SIZE = this->ROWS*this->COLUMNS*sizeof(T);
    int* mutex = 0;
    T *dmin, *da;
    T *minValue = new T;
  
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&dmin, sizeof(T)));
    gpuErrchk(hipMalloc((void**)&mutex, sizeof(int)));
    if (std::numeric_limits<T>::has_infinity){
        const T inf = std::numeric_limits<T>::infinity();
        gpuErrchk(hipMemset(dmin, inf , sizeof(T)));
    }
    else {
        return NULL;
    }
    gpuErrchk(hipMemset(mutex, 0, sizeof(int)));
    gpuErrchk(hipMemcpy(da, this->data, SIZE, hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(dmin, min, sizeof(T), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)this->ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , min(ceil((float)this->COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    // minGPU<<<blocksPerGrid, threadsPerBlock, ceil(((this->ROWS*this->COLUMNS)/(blocksPerGrid.x*THREADS_PER_BLOCK_DIM*blocksPerGrid.y*THREADS_PER_BLOCK_DIM+1))*256*sizeof(T))>>>(da, dmin, this->ROWS, this->COLUMNS, mutex);
    float sharedMemorySize = (float)(this->ROWS*this->COLUMNS)/(float)(carre(THREADS_PER_BLOCK_DIM)* blocksPerGrid.x * blocksPerGrid.y);
    minGPU<<<blocksPerGrid, threadsPerBlock, ceil(sharedMemorySize)*carre(THREADS_PER_BLOCK_DIM)*sizeof(T)>>>(da, dmin, this->ROWS, this->COLUMNS, mutex);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(minValue,  dmin, sizeof(T), hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(dmin));
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(mutex));    
    return *minValue;
}


template <typename T>
T GpuMatrix<T>::maxGpuMatrix(void){
    const size_t SIZE = this->ROWS*this->COLUMNS*sizeof(T);
    int* mutex = 0;
    T *dmax, *da;
    T *maxValue = new T;
  
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&dmax, sizeof(T)));
    gpuErrchk(hipMalloc((void**)&mutex, sizeof(int)));
    if (std::numeric_limits<T>::has_infinity){
        const T inf = (T)(-1) * std::numeric_limits<T>::infinity();
        gpuErrchk(hipMemset(dmax, 0 , sizeof(T)));
    }
    else {
        return NULL;
    }
    gpuErrchk(hipMemset(mutex, 0, sizeof(int)));
    gpuErrchk(hipMemcpy(da, this->data, SIZE, hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(dmin, min, sizeof(T), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)this->ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , min(ceil((float)this->COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    std::cout << ceil((float)this->ROWS/(float)THREADS_PER_BLOCK_DIM) << std::endl;

    // minGPU<<<blocksPerGrid, threadsPerBlock, ceil(((this->ROWS*this->COLUMNS)/(blocksPerGrid.x*THREADS_PER_BLOCK_DIM*blocksPerGrid.y*THREADS_PER_BLOCK_DIM+1))*256*sizeof(T))>>>(da, dmin, this->ROWS, this->COLUMNS, mutex);
    float sharedMemorySize = (float)(this->ROWS*this->COLUMNS)/(float)(carre(THREADS_PER_BLOCK_DIM)* blocksPerGrid.x * blocksPerGrid.y);
    maxGPU<<<blocksPerGrid, threadsPerBlock, ceil(sharedMemorySize)*carre(THREADS_PER_BLOCK_DIM)*sizeof(T)>>>(da, dmax, this->ROWS, this->COLUMNS, mutex);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(maxValue,  dmax, sizeof(T), hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(dmax));
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(mutex));    
    return *maxValue;
}


template <typename T> 
T GpuMatrix<T>::meanGpuMatrix(void){
    const size_t SIZE = this->ROWS*this->COLUMNS*sizeof(T);
    int* mutex = 0;
    T *dmean, *da;
    T *meanValue = new T;
  
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&dmean, sizeof(T)));
    gpuErrchk(hipMalloc((void**)&mutex, sizeof(int)));

    gpuErrchk(hipMemset(mutex, 0, sizeof(int)));
    gpuErrchk(hipMemset(dmean, 0, sizeof(T)));

    gpuErrchk(hipMemcpy(da, this->data, SIZE, hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(dmin, min, sizeof(T), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)this->ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , min(ceil((float)this->COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    // minGPU<<<blocksPerGrid, threadsPerBlock, ceil(((this->ROWS*this->COLUMNS)/(blocksPerGrid.x*THREADS_PER_BLOCK_DIM*blocksPerGrid.y*THREADS_PER_BLOCK_DIM+1))*256*sizeof(T))>>>(da, dmin, this->ROWS, this->COLUMNS, mutex);
    float sharedMemorySize = (float)(this->ROWS*this->COLUMNS)/(float)(carre(THREADS_PER_BLOCK_DIM)* blocksPerGrid.x * blocksPerGrid.y);
    meanGPU<<<blocksPerGrid, threadsPerBlock, ceil(sharedMemorySize)*carre(THREADS_PER_BLOCK_DIM)*sizeof(T)>>>(da, dmean, this->ROWS, this->COLUMNS, mutex);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(meanValue,  dmean, sizeof(T), hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(dmean));
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(mutex));    
    std::cout << *meanValue << std::endl;
    return *(meanValue)/(this->ROWS*this->COLUMNS);
}



// int main(void){
//     double sum;
//     struct Matrix<double> matrix= Matrix<double>{10000, 10000, new double[10000*10000]};
//     for (unsigned int i = 0; i<matrix.ROWS*matrix.COLUMNS; i++){
//         matrix.data[i] = (rand() % 100)+5;
//         sum += matrix.data[i];
//         // std::cout << "Value " << i << " : " << matrix.data[i] << " ---" << std::flush;
//     }
//     double minGPU = maxGPUMatrixFunction(matrix);
//     std::cout << "Max GPU : " << minGPU << std::endl; 
//     std::cout << "Moyenne CPU : " << sum/(matrix.COLUMNS*matrix.ROWS) << std::endl;

//     delete [] matrix.data;
//     return 0;
// }
