#include "hip/hip_runtime.h"
// Must be done... 
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h> 
#include <string>
#include <sstream>
#include <cstdlib>
#include <limits>

// #include "../../GPUOperations.h"
// #include "../initialisation/initialisation.cuh"
#include "statisticOperationsKernel.cuh"
#include "../../GpuMatrix.hpp"
#include "../generalInformation/generalInformation.hpp"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define assertm(exp, msg) assert(((void)msg, exp))
#define THREADS_PER_BLOCK_DIM 16
#define carre(x) (x*x)
#define minHost(a, b) (((a) < (b)) ? (a) : (b))

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
      std::cerr << hipGetErrorString(code) << " file : " << file << " line : " <<  line << std::endl;
      if (abort) { exit(code); }
    }
}

template <typename T> 
T GpuMatrix<T>::minGpuMatrix(void){
    const size_t SIZE = this->ROWS*this->COLUMNS*sizeof(T);
    int* mutex = 0;
    T *dmin, *da;
    T *minValue = new T;
  
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&dmin, sizeof(T)));
    gpuErrchk(hipMalloc((void**)&mutex, sizeof(int)));
    if (std::numeric_limits<T>::has_infinity){
        const T max = std::numeric_limits<T>::max();
        gpuErrchk(hipMemset(dmin, max , sizeof(T)));
    }
    else {
        exit(1);
    }
    gpuErrchk(hipMemset(mutex, 0, sizeof(int)));
    gpuErrchk(hipMemcpy(da, this->data, SIZE, hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(dmin, min, sizeof(T), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(minHost(ceil((float)this->ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , minHost(ceil((float)this->COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    // minGPU<<<blocksPerGrid, threadsPerBlock, ceil(((this->ROWS*this->COLUMNS)/(blocksPerGrid.x*THREADS_PER_BLOCK_DIM*blocksPerGrid.y*THREADS_PER_BLOCK_DIM+1))*256*sizeof(T))>>>(da, dmin, this->ROWS, this->COLUMNS, mutex);
    float sharedMemorySize = (float)(this->ROWS*this->COLUMNS)/(float)(carre(THREADS_PER_BLOCK_DIM)* blocksPerGrid.x * blocksPerGrid.y);
    minGPU<<<blocksPerGrid, threadsPerBlock, ceil(sharedMemorySize)*carre(THREADS_PER_BLOCK_DIM)*sizeof(T)>>>(da, dmin, this->ROWS, this->COLUMNS, mutex);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(minValue,  dmin, sizeof(T), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(dmin));
    gpuErrchk(hipFree(da));
    // std::cout << "MinGPU will be " << *minValue << std::endl;
    gpuErrchk(hipFree(mutex));    
    return *minValue;
}


template <typename T>
T GpuMatrix<T>::maxGpuMatrix(void){
    const size_t SIZE = this->ROWS*this->COLUMNS*sizeof(T);
    int* mutex = 0;
    T *dmax, *da;
    T *maxValue = new T;
  
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&dmax, sizeof(T)));
    gpuErrchk(hipMalloc((void**)&mutex, sizeof(int)));
    if (std::numeric_limits<T>::has_infinity){
        const T min = std::numeric_limits<T>::min();
        gpuErrchk(hipMemset(dmax, min , sizeof(T)));
    }
    else {
        exit(1);
    }
    gpuErrchk(hipMemset(mutex, 0, sizeof(int)));
    gpuErrchk(hipMemcpy(da, this->data, SIZE, hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(dmin, min, sizeof(T), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(minHost(ceil((float)this->ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , minHost(ceil((float)this->COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    // minGPU<<<blocksPerGrid, threadsPerBlock, ceil(((this->ROWS*this->COLUMNS)/(blocksPerGrid.x*THREADS_PER_BLOCK_DIM*blocksPerGrid.y*THREADS_PER_BLOCK_DIM+1))*256*sizeof(T))>>>(da, dmin, this->ROWS, this->COLUMNS, mutex);
    float sharedMemorySize = (float)(this->ROWS*this->COLUMNS)/(float)(carre(THREADS_PER_BLOCK_DIM)* blocksPerGrid.x * blocksPerGrid.y);
    maxGPU<<<blocksPerGrid, threadsPerBlock, ceil(sharedMemorySize)*carre(THREADS_PER_BLOCK_DIM)*sizeof(T)>>>(da, dmax, this->ROWS, this->COLUMNS, mutex);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(maxValue,  dmax, sizeof(T), hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(dmax));
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(mutex));    
    return *maxValue;
}


template <typename T> 
T GpuMatrix<T>::meanGpuMatrix(void){
    const size_t SIZE = this->ROWS*this->COLUMNS*sizeof(T);
    int* mutex = 0;
    T *dmean, *da;
    T *meanValue = new T;
  
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&dmean, sizeof(T)));
    gpuErrchk(hipMalloc((void**)&mutex, sizeof(int)));

    gpuErrchk(hipMemset(mutex, 0, sizeof(int)));
    gpuErrchk(hipMemset(dmean, 0, sizeof(T)));

    gpuErrchk(hipMemcpy(da, this->data, SIZE, hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(dmin, min, sizeof(T), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(minHost(ceil((float)this->ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , minHost(ceil((float)this->COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    // minGPU<<<blocksPerGrid, threadsPerBlock, ceil(((this->ROWS*this->COLUMNS)/(blocksPerGrid.x*THREADS_PER_BLOCK_DIM*blocksPerGrid.y*THREADS_PER_BLOCK_DIM+1))*256*sizeof(T))>>>(da, dmin, this->ROWS, this->COLUMNS, mutex);
    float sharedMemorySize = (float)(this->ROWS*this->COLUMNS)/(float)(carre(THREADS_PER_BLOCK_DIM)* blocksPerGrid.x * blocksPerGrid.y);
    meanGPU<<<blocksPerGrid, threadsPerBlock, ceil(sharedMemorySize)*carre(THREADS_PER_BLOCK_DIM)*sizeof(T)>>>(da, dmean, this->ROWS, this->COLUMNS, mutex);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(meanValue,  dmean, sizeof(T), hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(dmean));
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(mutex));    
    return *(meanValue)/(this->ROWS*this->COLUMNS);
}


