#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>       /* ceil */
#include <functional>
#include <nvfunctional>
#include <assert.h>

// #include "../initialisation/initialisation.cuh"
// #include "../../GPUOperations.h"
#include "arithmeticOperationsKernel.cuh"


// It would be ideal to transfert data while executing kernel device operations

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define EXIT_SUCESS 0 
#define __GPU_EXP__
#define THREADS_PER_BLOCK_DIM 16
#define min(a, b) (((a)>(b)) ? (b) : (a))
#define carre(x) (x*x)
#define assertm(exp, msg) assert(((void)msg, exp))
#define functionDeviceHost(func) (__device__ __host__  func)

// const hipDeviceProp_t deviceProps;
// typedef __device__ __host__ auto lambdaExpression;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
      std::cerr << hipGetErrorString(code) << " file : " << file << " line : " <<  line << std::endl;
      if (abort) { exit(code); }
    }
}

/**************************************************************************************************
Kernel functions in cuh file
************************************************************************************************/



template <typename T>
struct Matrix {
    size_t ROWS;
    size_t COLUMNS;
    T* data;
}; 
// const hipDeviceProp_t deviceProps;
struct typeProps{
    int maxGridSize;
} variableProps = {65355};

template <typename T>
__host__ Matrix<T> add(Matrix<T> a, Matrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS==b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*b.COLUMNS*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min((a.ROWS/THREADS_PER_BLOCK_DIM), (variableProps.maxGridSize)) , min((a.COLUMNS/THREADS_PER_BLOCK_DIM), (variableProps.maxGridSize)));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    addGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, a.ROWS, a.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));

    Matrix<T> toReturn;
    toReturn.ROWS = a.ROWS;
    toReturn.COLUMNS = a.COLUMNS;
    toReturn.data = hc;
    return toReturn;

}

template <typename T>
__host__ Matrix<T> substract(Matrix<T> a, Matrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS==b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*b.COLUMNS*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    substractGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, a.ROWS, a.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));

    Matrix<T> toReturn;
    toReturn.ROWS = a.ROWS;
    toReturn.COLUMNS = a.COLUMNS;
    toReturn.data = hc;
    return toReturn;

}

template <typename T>
__host__ Matrix<T> multiply(Matrix<T> a, Matrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS==b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*b.COLUMNS*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    multiplyGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, a.ROWS, a.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));
    gpuErrchk(hipHostFree(hc));

    Matrix<T> toReturn;
    toReturn.ROWS = a.ROWS;
    toReturn.COLUMNS = a.COLUMNS;
    toReturn.data = hc;
    return toReturn;

}

template <typename T>
__host__ Matrix<T> scalarMultiply(T a, Matrix<T> b){
    // assertm((a.ROWS==b.ROWS && a.COLUMNS*b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = b.ROWS*b.COLUMNS*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, sizeof(T)));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a, sizeof(T), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    scalarMultiplyGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, a.ROWS, a.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));

    Matrix<T> toReturn;
    toReturn.ROWS = a.ROWS;
    toReturn.COLUMNS = a.COLUMNS;
    toReturn.data = hc;
    return toReturn;

}

template <typename T>
__host__ Matrix<T> divide(Matrix<T> a, Matrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS == b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*a.COLUMNS*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    divideGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, a.ROWS, a.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));

    Matrix<T> toReturn;
    toReturn.ROWS = a.ROWS;
    toReturn.COLUMNS = a.COLUMNS;
    toReturn.data = hc;
    return toReturn;

}
// template <typename T, typename F>
// __host__ __device__ F lambdaGPU(T x, const nvstd::function<F(T&)> func){
//     return func(x);
// }


template <typename T, typename F>
__host__ Matrix<F> applyLambdaToElementMatrix(Matrix<T> a, F lambdaFunction){
    
    #ifndef __CUDACC_EXTENDED_LAMBDA__
    #error "please compile with --expt-extended-lambda add it to make file"
    #endif

    const size_t SIZE_T = a.ROWS*a.COLUMNS*sizeof(T);
    const size_t SIZE_F = a.ROWS*a.COLUMNS*sizeof(F);
    T* d_a;
    F* d_result, *result;
    // int x = 2;
    // lambdaGPU(x , lambdaFunction);
    // __device__ auto lambdaFunctionOnGPU = lambdaFunction; // need of --expt-extended-lambda
    
    // std::cout  << lambdaFunctionOnGPU(x) << std::endl;
    gpuErrchk(hipMalloc((void**)&d_a, SIZE_T));
    gpuErrchk(hipMalloc((void**)&d_result, SIZE_F));
    gpuErrchk(hipHostAlloc((void**)&result, SIZE_F, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(d_a, a, SIZE_T, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_result, result, SIZE_F, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);
    // std::cout << *(a + 20000) << std::endl;

    applyLambdaToElementMatrixGPU<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_result, lambdaFunction, a.ROWS, a.COLUMNS);
    // applyLambdaToElementMatrixGPU<<<(1, 1), (1, 1) >>>(d_a, d_result, lambdaFunctionOnGPU, ROWS, COLUMN);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(result, d_result, SIZE_F, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_a));
    gpuErrchk(hipFree(d_result));

    // return result;c
    Matrix<T> toReturn;
    toReturn.ROWS = a.ROWS;
    toReturn.COLUMNS = a.COLUMNS;
    toReturn.data = result;
    return toReturn;
}

int main(void){
    struct Matrix<double> matrix= Matrix<double>{16, 16, new double[16*16]};

    for (unsigned int i = 0; i<matrix.ROWS*matrix.COLUMNS; i++){
        matrix.data[i] = 2;
        // std::cout << "Value " << i << " : " << matrix.data[i] << " ---" << std::flush;
    }

    struct Matrix<double> result = applyLambdaToElementMatrix<double>(matrix, carre);
    // gpuPrint(matrix, 10, 10);
    // gpuPrint(result, 10, 10);
    delete [] matrix.data;
    // delete [] matrixR.data;
    return 0;
}