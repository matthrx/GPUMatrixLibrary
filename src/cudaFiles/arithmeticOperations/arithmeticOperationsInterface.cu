#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>       /* ceil */
#include <functional>
#include <nvfunctional>

#include "arithmeticOperationsKernel.cuh"


// It would be ideal to transfert data while executing kernel device operations

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define THREAD_PER_BLOCK 32
#define EXIT_SUCESS 0 
#define __GPU_EXP__

// typedef __device__ __host__ auto lambdaExpression;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
      std::cerr << hipGetErrorString(code) << " file : " << file << " line : " <<  line << std::endl;
      if (abort) { exit(code); }
    }
}

/**************************************************************************************************
Kernel functions in cuh file
************************************************************************************************/



template <typename T>
__host__ void addHost(T* a, T* b, T* c, int ROWS, int COLUMN){
    const size_t SIZE = ROWS*COLUMN*sizeof(T);
    T *da, *db, *dc;
 
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));

    gpuErrchk(hipMemcpy(da, a, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dc, c, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(16, 16);
    dim3 threadsPerBlock(THREAD_PER_BLOCK, THREAD_PER_BLOCK);

    addGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, ROWS, COLUMN);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(c,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));

}

// template <typename T, typename F>
// __host__ __device__ F lambdaGPU(T x, const nvstd::function<F(T&)> func){
//     return func(x);
// }

template <typename T, typename F>
__host__ void applyLambdaToElementMatrix(const T* a, nvstd::function<F(T&)> lambdaFunction, int ROWS, int COLUMN){
    
    #ifndef __CUDACC_EXTENDED_LAMBDA__
    #error "please compile with --expt-extended-lambda"
    #endif

    const size_t SIZE_T = ROWS*COLUMN*sizeof(T);
    const size_t SIZE_F = ROWS*COLUMN*sizeof(F);
    T* d_a;
    F* d_result;

    F* result = new F[ROWS*COLUMN];
    // int x = 2;
    // lambdaGPU(x , lambdaFunction);
    // __device__ auto lambdaFunctionOnGPU = lambdaFunction; // need of --expt-extended-lambda
    
    // std::cout  << lambdaFunctionOnGPU(x) << std::endl;
    gpuErrchk(hipMalloc((void**)&d_a, SIZE_T));
    gpuErrchk(hipMalloc((void**)&d_result, SIZE_F));
    
    gpuErrchk(hipMemcpy(d_a, a, SIZE_T, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_result, result, SIZE_F, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(ceil(ROWS/THREAD_PER_BLOCK), ceil(COLUMN/THREAD_PER_BLOCK));

    // dim3 blocksPerGrid(16, 16);
    dim3 threadsPerBlock(THREAD_PER_BLOCK, THREAD_PER_BLOCK);
    // std::cout << *(a + 20000) << std::endl;

    applyLambdaToElementMatrixGPU<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_result, lambdaFunction, ROWS, COLUMN);
    // applyLambdaToElementMatrixGPU<<<(1, 1), (1, 1) >>>(d_a, d_result, lambdaFunctionOnGPU, ROWS, COLUMN);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(result, d_result, SIZE_T, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_a));
    gpuErrchk(hipFree(d_result));

    std::cout << *(result + 20000) << std::endl;
    // return result;c
}

// double carre(int x){
//     return pow(x, 2);
// }

int main(void){
    const long int ROWS_MATRIX = 32000;
    const long int COLUMNS_MATRIX = 32000;
    int* matrix = new int[ROWS_MATRIX*COLUMNS_MATRIX];

    auto carre = [] __GPU_EXP__ (int x){return pow(x,2);};
    for (unsigned int i = 0; i < (ROWS_MATRIX*COLUMNS_MATRIX); i ++){
        *(matrix + i) = reinterpret_cast<int>(rand()%10);
    }
    std::cout << *(matrix+12000) << std::endl;

    applyLambdaToElementMatrix<int, double>(matrix, carre, ROWS_MATRIX, COLUMNS_MATRIX);
    std::cout << "Stop..." << std::flush;
    for (unsigned int i = 0; i < (ROWS_MATRIX*COLUMNS_MATRIX); i ++){
        *(matrix + i) = carre(*(matrix + i));
    }
    std::cout << *(matrix+120000) << std::endl;
    // std::cout << std::to_string(carre) << std::endl;
    delete [] matrix;
    return EXIT_SUCESS;
}
