#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>       /* ceil */
#include <functional>
#include <nvfunctional>
#include <assert.h>

#include "../initialisation/initialisation.cuh"
#include "../../GPUOperations.h"
#include "arithmeticOperationsKernel.cuh"


// It would be ideal to transfert data while executing kernel device operations

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define EXIT_SUCESS 0 
#define __GPU_EXP__
#define THREADS_PER_BLOCK_DIM 16
#define min(a, b) (((a)>(b)) ? (b) : (a))
#define carre(x) (x*x)
#define assertm(exp, msg) assert(((void)msg, exp))

const hipDeviceProp_t deviceProps;
// typedef __device__ __host__ auto lambdaExpression;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
      std::cerr << hipGetErrorString(code) << " file : " << file << " line : " <<  line << std::endl;
      if (abort) { exit(code); }
    }
}

/**************************************************************************************************
Kernel functions in cuh file
************************************************************************************************/

template <typename T>
struct Matrix {
    size_t ROWS;
    size_t COLUMNS;
    T* data;
}; 

template <typename T>
__host__ Matrix<T> add(Matrix<T> a, Matrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS*b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*b.COLUMN*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])) , min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    addGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, a.ROWS, a.COLUMN);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));
    gpuErrchk(hipHostFree(hc));

    return Matrix<T>{
        a.ROWS; a.COLUMNS; hc
    };

}

template <typename T>
__host__ Matrix<T> substract(Matrix<T> a, Matrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS*b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*b.COLUMN*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])) , min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    substractGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, ROWS, COLUMN);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));
    gpuErrchk(hipHostFree(hc));

    return Matrix<T>{
        a.ROWS; a.COLUMNS; hc
    };

}

template <typename T>
__host__ Matrix<T> multiply(Matrix<T> a, Matrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS*b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*b.COLUMN*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])) , min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    multiplyGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, ROWS, COLUMN);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));
    gpuErrchk(hipHostFree(hc));

    return Matrix<T>{
        a.ROWS; a.COLUMNS; hc
    };

}

template <typename T>
__host__ Matrix<T> scalarMultiply(T a, Matrix<T> b){
    // assertm((a.ROWS==b.ROWS && a.COLUMNS*b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*b.COLUMN*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, sizeof(T)));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a, sizeof(T), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])) , min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    scalarMultiplyGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, ROWS, COLUMN);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));
    gpuErrchk(hipHostFree(hc));

    return Matrix<T>{
        a.ROWS; a.COLUMNS; hc
    };

}

template <typename T>
__host__ Matrix<T> divide(Matrix<T> a, Matrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS*b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*b.COLUMN*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])) , min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    divideGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, ROWS, COLUMN);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));
    gpuErrchk(hipHostFree(hc));

    return Matrix<T>{
        a.ROWS; a.COLUMNS; hc
    };

}
// template <typename T, typename F>
// __host__ __device__ F lambdaGPU(T x, const nvstd::function<F(T&)> func){
//     return func(x);
// }

template <typename T, typename F>
__host__ Matrix<T> applyLambdaToElementMatrix(const T* a, nvstd::function<F(T&)> lambdaFunction, int ROWS, int COLUMN){
    
    #ifndef __CUDACC_EXTENDED_LAMBDA__
    #error "please compile with --expt-extended-lambda add it to make file"
    #endif

    const size_t SIZE_T = a.ROWS*a.COLUMN*sizeof(T);
    const size_t SIZE_F = a.ROWS*a.COLUMN*sizeof(F);
    T* d_a;
    F* d_result, *result;
    // int x = 2;
    // lambdaGPU(x , lambdaFunction);
    // __device__ auto lambdaFunctionOnGPU = lambdaFunction; // need of --expt-extended-lambda
    
    // std::cout  << lambdaFunctionOnGPU(x) << std::endl;
    gpuErrchk(hipMalloc((void**)&d_a, SIZE_T));
    gpuErrchk(hipMalloc((void**)&d_result, SIZE_F));
    gpuErrchk(hipHostAlloc((void**)&result, SIZE_F, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(d_a, a, SIZE_T, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_result, result, SIZE_F, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])) , min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);
    // std::cout << *(a + 20000) << std::endl;

    applyLambdaToElementMatrixGPU<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_result, lambdaFunction, ROWS, COLUMN);
    // applyLambdaToElementMatrixGPU<<<(1, 1), (1, 1) >>>(d_a, d_result, lambdaFunctionOnGPU, ROWS, COLUMN);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(result, d_result, SIZE_F, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_a));
    gpuErrchk(hipFree(d_result));
    gpuErrchk(hipHostFree(result));

    // return result;c
    return Matrix<T> {
        a.ROWS; a.COLUMNS; result
    };
}

// double carre(int x){
//     return pow(x, 2);
// }
