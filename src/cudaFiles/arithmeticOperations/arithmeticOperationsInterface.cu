#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>       /* ceil */
#include <functional>
#include <nvfunctional>
#include <assert.h>

// #include "../initialisation/initialisation.cuh"
// #include "../../GPUOperations.h"

#include "arithmeticOperationsKernel.cuh"
#include "../generalInformation/generalInformation.h"
#include "../../GpuMatrix.h"

// It would be ideal to transfert data while executing kernel device operations

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define EXIT_SUCESS 0 
#define __GPU_EXP__
#define THREADS_PER_BLOCK_DIM 16
#define min(a, b) (((a)>(b)) ? (b) : (a))
#define carre(x) (x*x)
#define assertm(exp, msg) assert(((void)msg, exp))
#define functionDeviceHost(func) (__device__ __host__  func)

// const hipDeviceProp_t deviceProps;
// typedef __device__ __host__ auto lambdaExpression;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
      std::cerr << hipGetErrorString(code) << " file : " << file << " line : " <<  line << std::endl;
      if (abort) { exit(code); }
    }
}

/**************************************************************************************************
Kernel functions in cuh file
************************************************************************************************/

template <typename T>
GpuMatrix<T> GpuMatrix<T>::add(GpuMatrix<T> a, GpuMatrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS==b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*b.COLUMNS*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    addGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, a.ROWS, a.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));
    GpuMatrix<T> toReturn = GpuMatrix<T>(a.ROWS, a.COLUMNS, hc);
    return toReturn;

}

template <typename T>
GpuMatrix<T> GpuMatrix<T>::substract(GpuMatrix<T> a, GpuMatrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS==b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*b.COLUMNS*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    substractGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, a.ROWS, a.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));

    GpuMatrix<T> toReturn = GpuMatrix<T>(a.ROWS, a.COLUMNS, hc);
    return toReturn;


}

template <typename T>
GpuMatrix<T> GpuMatrix<T>::multiply(GpuMatrix<T> a, GpuMatrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS==b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*b.COLUMNS*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    multiplyGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, a.ROWS, a.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));

    GpuMatrix<T> toReturn = GpuMatrix<T>(a.ROWS, b.COLUMNS, hc);
    return toReturn;


}

template <typename T>
GpuMatrix<T> GpuMatrix<T>::scalarMultiply(T a, GpuMatrix<T> b){
    // assertm((a.ROWS==b.ROWS && a.COLUMNS*b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = b.ROWS*b.COLUMNS*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, sizeof(T)));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, &a, sizeof(T), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)b.ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , min(ceil((float)b.COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    scalarMultiplyGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, b.ROWS, b.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));

    GpuMatrix<T> toReturn = GpuMatrix<T>(b.ROWS, b.COLUMNS, hc);
    return toReturn;


}

template <typename T>
 GpuMatrix<T> GpuMatrix<T>::divide(GpuMatrix<T> a, GpuMatrix<T> b){
    assertm((a.ROWS==b.ROWS && a.COLUMNS == b.COLUMNS), "Error incompatible dimensions, can't apply the operator");
    const size_t SIZE = a.ROWS*a.COLUMNS*sizeof(T);
    T *da, *db, *dc, *hc;
    
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&dc, SIZE));
    gpuErrchk(hipHostAlloc((void**)&hc, SIZE, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    divideGPU<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, a.ROWS, a.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(hc,  dc, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(dc));

    GpuMatrix<T> toReturn = GpuMatrix<T>(a.ROWS, a.COLUMNS, hc);
    return toReturn;


}
// template <typename T, typename F>
// __host__ __device__ F lambdaGPU(T x, const nvstd::function<F(T&)> func){
//     return func(x);
// }


template <typename T, typename F>
 GpuMatrix<F> applyLambdaToElementMatrix(GpuMatrix<T> a, F lambdaFunction){
    
    // #ifndef __CUDACC_EXTENDED_LAMBDA__
    // #error "please compile with --expt-extended-lambda add it to make file"
    // #endif

    const size_t SIZE_T = a.ROWS*a.COLUMNS*sizeof(T);
    const size_t SIZE_F = a.ROWS*a.COLUMNS*sizeof(F);
    T* d_a;
    F* d_result, *result;
    // int x = 2;
    // lambdaGPU(x , lambdaFunction);
    // __device__ auto lambdaFunctionOnGPU = lambdaFunction; // need of --expt-extended-lambda
    
    // std::cout  << lambdaFunctionOnGPU(x) << std::endl;
    gpuErrchk(hipMalloc((void**)&d_a, SIZE_T));
    gpuErrchk(hipMalloc((void**)&d_result, SIZE_F));
    gpuErrchk(hipHostAlloc((void**)&result, SIZE_F, hipHostMallocDefault));

    gpuErrchk(hipMemcpy(d_a, a, SIZE_T, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_result, result, SIZE_F, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);
    // std::cout << *(a + 20000) << std::endl;

    applyLambdaToElementMatrixGPU<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_result, lambdaFunction, a.ROWS, a.COLUMNS);
    // applyLambdaToElementMatrixGPU<<<(1, 1), (1, 1) >>>(d_a, d_result, lambdaFunctionOnGPU, ROWS, COLUMN);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(result, d_result, SIZE_F, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_a));
    gpuErrchk(hipFree(d_result));

    // return result;c
    GpuMatrix<F> toReturn = GpuMatrix<F>(a.ROWS, a.COLUMNS, result);
    return toReturn;

}

// int main(void){
//     struct Matrix<double> matrix= Matrix<double>{16, 16, new double[16*16]};

//     for (unsigned int i = 0; i<matrix.ROWS*matrix.COLUMNS; i++){
//         matrix.data[i] = 2;
//         // std::cout << "Value " << i << " : " << matrix.data[i] << " ---" << std::flush;
//     }

//     struct Matrix<double> result = applyLambdaToElementMatrix<double>(matrix, carre);
//     // gpuPrint(matrix, 10, 10);
//     // gpuPrint(result, 10, 10);
//     delete [] matrix.data;
//     // delete [] matrixR.data;
//     return 0;
// }

// template float twice<float>(float original)
// template GpuMatrix<double> GpuMatrix<double>::add(GpuMatrix<double>, GpuMatrix<double>);