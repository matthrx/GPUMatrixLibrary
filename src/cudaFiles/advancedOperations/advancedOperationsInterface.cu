#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h> 
#include <string>
#include <sstream>

#include "advancedOperationsKernel.cuh"
#include "../../GpuMatrix.hpp"
#include "../generalInformation/generalInformation.hpp"
// #include "../../GPUOperations.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define assertm(exp, msg) assert(((void)msg, exp))
#define GET_VARIABLE_NAME(a) (#a)
#define EXIT_SUCESS 0 
#define __GPU_EXP__
#define THREADS_PER_BLOCK_DIM 16
#define min(a, b) (((a)>(b)) ? (b) : (a))
#define carre(x) (x*x)

// typedef __device__ __host__ auto lambdaExpression;
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
      std::cerr << hipGetErrorString(code) << " file : " << file << " line : " <<  line << std::endl;
      if (abort) { exit(code); }
    }
}
template <class T>
GpuMatrix<T> GpuMatrix<T>::transpose(void){
    const size_t SIZE = this->ROWS*this->COLUMNS*sizeof(T);
    T *da, *dataTranspose, *d_dataTranspose;


    gpuErrchk(hipHostAlloc((void**)&dataTranspose, SIZE, hipHostMallocDefault));
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&d_dataTranspose, SIZE));

    gpuErrchk(hipMemcpy(da, this->data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)this->ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , min(ceil((float)this->COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    float sharedMemorySize = (float)(this->ROWS*this->COLUMNS)/(float)(carre(THREADS_PER_BLOCK_DIM)* blocksPerGrid.x * blocksPerGrid.y);
    transposeKernel<<<blocksPerGrid, threadsPerBlock, ceil(sharedMemorySize)*carre(THREADS_PER_BLOCK_DIM)*sizeof(T)>>>(da, d_dataTranspose, this->ROWS, this->COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(dataTranspose,  d_dataTranspose, SIZE, hipMemcpyDeviceToHost));
    
    // gpuErrchk(hipHostFree(dataTranpose));

    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(d_dataTranspose));

    GpuMatrix<T> toReturn = GpuMatrix<T>(this->ROWS, this->COLUMNS, dataTranspose);
    // toReturn.ROWS = a.COLUMNS;
    // toReturn.COLUMNS = a.ROWS;
    // toReturn.data = dataTranspose;
    return toReturn;
}

template <typename T>
GpuMatrix<T> GpuMatrix<T>::dot(GpuMatrix<T> b){
    std::ostringstream alertMessage;
    alertMessage << "Error : Those matrixes can't be multiplied check their dimensions \n In product A.B where A is "<< GET_VARIABLE_NAME(this) << "and B is " << GET_VARIABLE_NAME(b) << " : dim(A)=[" << this->ROWS
    << "," << this->COLUMNS << "] & dim(B) = ["<< b.ROWS << "," << b.COLUMNS << "]";
    assertm(this->ROWS == b.COLUMNS && this->COLUMNS== b.ROWS, alertMessage.str());
    const size_t SIZE = this->ROWS*b.COLUMNS*sizeof(T);
    T *da, *db, *d_dataProduct, *dataProduct;

    gpuErrchk(hipHostAlloc((void**)&dataProduct, SIZE, hipHostMallocDefault));
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&d_dataProduct, SIZE));

    gpuErrchk(hipMemcpy(da, this->data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)this->ROWS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[0]) , min(ceil((float)this->COLUMNS/(float)THREADS_PER_BLOCK_DIM), deviceProps.maxGridSize[1]));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    dotKernel<<<blocksPerGrid, threadsPerBlock>>>(da, db, d_dataProduct, this->ROWS, b.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(dataProduct,  d_dataProduct, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(d_dataProduct));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(da));
    
    GpuMatrix<T> toReturn = GpuMatrix<T>(this->ROWS, b.COLUMNS, dataProduct);
    // toReturn.ROWS = a.ROWS;
    // toReturn.COLUMNS = a.COLUMNS;
    // toReturn.data = dataProduct;
    return toReturn;
}

// int main(void){
//     struct Matrix<double> matrix= Matrix<double>{16, 16, new double[16*16]};
//     struct Matrix<double> matrixR= Matrix<double>{16, 16, new double[16*16]};

//     for (unsigned int i = 0; i<matrix.ROWS*matrix.COLUMNS; i++){
//         matrix.data[i] = 1;
//         matrixR.data[i] = 2;
//         // std::cout << "Value " << i << " : " << matrix.data[i] << " ---" << std::flush;
//     }
//     struct Matrix<double> result = dotInterface(matrix, matrixR);
//     gpuPrint(matrix, 10, 10);
//     gpuPrint(result, 10, 10);
//     delete [] matrix.data;
//     delete [] matrixR.data;
//     return 0;
// }