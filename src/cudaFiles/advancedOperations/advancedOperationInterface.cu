#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h> 
#include <string>
#include <sstream>

#include "advancedOperationsKernel.cuh"
#include "../generalInformation/generalInformation.cuh"
// #include "../../GPUOperations.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define assertm(exp, msg) assert(((void)msg, exp))
#define GET_VARIABLE_NAME(a) (#a)
#define EXIT_SUCESS 0 
#define __GPU_EXP__
#define THREADS_PER_BLOCK_DIM 16
#define min(a, b) (((a)>(b)) ? (b) : (a))
#define carre(x) (x*x)


struct typeProps{
    int maxGridSize;
} variableProps = {65355};

// const hipDeviceProp_t deviceProps;
// typedef __device__ __host__ auto lambdaExpression;
// inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
// {
//     if (code != hipSuccess)
//     {
//       std::cerr << hipGetErrorString(code) << " file : " << file << " line : " <<  line << std::endl;
//       if (abort) { exit(code); }
//     }
// }
    
template <typename T>
__host__ Matrix<T> transposeInterface(Matrix<T> a){
    const size_t SIZE = a.ROWS*a.COLUMNS*sizeof(T);
    T *da, *dataTranspose, *d_dataTranspose;


    gpuErrchk(hipHostAlloc((void**)&dataTranspose, SIZE, hipHostMallocDefault));
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&d_dataTranspose, SIZE));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    float sharedMemorySize = (float)(a.ROWS*a.COLUMNS)/(float)(carre(THREADS_PER_BLOCK_DIM)* blocksPerGrid.x * blocksPerGrid.y);
    transpose<<<blocksPerGrid, threadsPerBlock, ceil(sharedMemorySize)*carre(THREADS_PER_BLOCK_DIM)*sizeof(T)>>>(da, d_dataTranspose, a.ROWS, a.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(dataTranspose,  d_dataTranspose, SIZE, hipMemcpyDeviceToHost));
    
    // gpuErrchk(hipHostFree(dataTranpose));

    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(d_dataTranspose));

    Matrix<T> toReturn;
    toReturn.ROWS = a.COLUMNS;
    toReturn.COLUMNS = a.ROWS;
    toReturn.data = dataTranspose;
    return toReturn;
}

template <typename T>
__host__ Matrix<T> dotInterface(Matrix<T> a, Matrix<T> b){
    std::ostringstream alertMessage;
    alertMessage << "Error : Those matrixes can't be multiplied check their dimensions \n In product A.B where A is "<< GET_VARIABLE_NAME(a) << "and B is " << GET_VARIABLE_NAME(b) << " : dim(A)=[" << a.ROWS
    << "," << a.COLUMNS << "] & dim(B) = ["<< b.ROWS << "," << b.COLUMNS << "]";
    assertm(a.ROWS == b.COLUMNS && a.COLUMNS== b.ROWS, alertMessage.str());
    const size_t SIZE = a.ROWS*b.COLUMNS*sizeof(T);
    T *da, *db, *d_dataProduct, *dataProduct;

    gpuErrchk(hipHostAlloc((void**)&dataProduct, SIZE, hipHostMallocDefault));
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&d_dataProduct, SIZE));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min(ceil((float)a.ROWS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize) , min(ceil((float)a.COLUMNS/(float)THREADS_PER_BLOCK_DIM), variableProps.maxGridSize));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    dot<<<blocksPerGrid, threadsPerBlock>>>(da, db, d_dataProduct, a.ROWS, b.COLUMNS);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(dataProduct,  d_dataProduct, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipFree(d_dataProduct));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(da));
    
    Matrix<T> toReturn;
    toReturn.ROWS = a.ROWS;
    toReturn.COLUMNS = a.COLUMNS;
    toReturn.data = dataProduct;
    return toReturn;
}

int main(void){
    struct Matrix<double> matrix= Matrix<double>{16, 16, new double[16*16]};
    struct Matrix<double> matrixR= Matrix<double>{16, 16, new double[16*16]};

    for (unsigned int i = 0; i<matrix.ROWS*matrix.COLUMNS; i++){
        matrix.data[i] = 1;
        matrixR.data[i] = 2;
        // std::cout << "Value " << i << " : " << matrix.data[i] << " ---" << std::flush;
    }
    struct Matrix<double> result = dotInterface(matrix, matrixR);
    gpuPrint(matrix, 10, 10);
    gpuPrint(result, 10, 10);
    delete [] matrix.data;
    delete [] matrixR.data;
    return 0;
}