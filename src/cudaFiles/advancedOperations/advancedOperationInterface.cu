#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h> 
#include <string>
#include <sstream>

#include "advancedOperationsKernel.cuh"
#include "../initialisation/initialisation.cuh"
#include "../../GPUOperations.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define assertm(exp, msg) assert(((void)msg, exp))
#define EXIT_SUCESS 0 
#define __GPU_EXP__
#define THREADS_PER_BLOCK_DIM 16
#define min(a, b) (((a)>(b)) ? (b) : (a))
#define carre(x) (x*x)


const hipDeviceProp_t deviceProps;
// typedef __device__ __host__ auto lambdaExpression;
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
      std::cerr << hipGetErrorString(code) << " file : " << file << " line : " <<  line << std::endl;
      if (abort) { exit(code); }
    }
}
    
template <typename T>
__host__ Matrix<T> transposeInterface(Matrix<T> a){
    const size_t SIZE = a.ROWS*a.COLUMN*sizeof(T);
    T *da, *dataTranspose, *d_dataTranspose;


    gpuErrchk(hipHostAlloc((void**)&dataTranspose, SIZE, hipHostMallocDefault));
    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&d_dataTranspose, SIZE));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])) , min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    transpose<<<blocksPerGrid, threadsPerBlock, ceil(((a.ROWS*a.COLUMNS)/(16*THREADS_PER_BLOCK_DIM*16*THREADS_PER_BLOCK_DIM))*256*sizeof(T)>)>>(da, d_dataTranspose, a.ROWS, a.COLUMN);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(dataTranpose,  d_dataTranspose, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipHostFree(dataTranpose));

    gpuErrchk(hipFree(da));
    gpuErrchk(hipFree(d_dataTranspose));

    Matrix finalMatrix{
        a.COLUMN; a.ROWS; dataTranpose;
    };
    return finalMatrix;
}

template <typename T>
__host__ Matrix<T> dotInterface(Matrix<T> a, Matrix<T> b){
    std::ostringstream alertMessage;
    alertMessage << "Error : Those matrixes can't be multiplied check their dimensions \n In product A.B where A is "<< #A "and B is " << #B << " : dim(A)=[" << a.ROWS
    << "," << a.COLUMNS << "] & dim(B) = ["<< b.ROWS << "," << b.COLUMNS << "]";
    assertm(a.ROWS == b.COLUMN && a.COLUMN == b.ROWS, alertMessage.str());
    const size_t SIZE = ROWS*COLUMN*sizeof(T);
    T *da, *db, *d_dataProduct, *dataProduct;


    gpuErrchk(hipHostAlloc((void**)&dataProduct SIZE, hipHostMallocDefault));

    gpuErrchk(hipMalloc((void**)&da, SIZE));
    gpuErrchk(hipMalloc((void**)&db, SIZE));
    gpuErrchk(hipMalloc((void**)&d_dataProduct, SIZE));

    gpuErrchk(hipMemcpy(da, a.data, SIZE, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, b.data, SIZE, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])) , min((ROWS/THREADS_PER_BLOCK_DIM), (deviceProps.maxGridSize[0])));
    dim3 threadsPerBlock(THREADS_PER_BLOCK_DIM, THREADS_PER_BLOCK_DIM);

    dot<<<blocksPerGrid, threadsPerBlock>>>(da, db, d_dataProduct, ROWS, COLUMN);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(dataProduct,  d_dataProduct, SIZE, hipMemcpyDeviceToHost));
    
    gpuErrchk(hipHostFree(dataProduct));
    gpuErrchk(hipFree(d_dataProduct));
    gpuErrchk(hipFree(db));
    gpuErrchk(hipFree(da));
    
    Matrix toReturn {
        a.ROWS b.COLUMN; dataProduct;
    };
    return toReturn;
}

