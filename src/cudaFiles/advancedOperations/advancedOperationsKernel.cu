#include "hip/hip_runtime.h"
#ifndef __ADVANCED_OPERATIONS_KERNEL__
#define __ADVANCED_OPERATIONS_KERNEL__

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>

template <typename T>
typedef struct {
    size_t rows;
    size_t columns;
    T* elements
} Matrix;


__global__ void transpose(const Matrix  __restrict__ a, Matrix b,) {
    unsigned int tidX = threadIdx.x + blockDim.x* blockIdx.x;
    unsigned int tidY = threadIdx.y + blockDim.y* blockIdx.y;
    unsigned long int stride = gridDim.x*blockDim.x + gridDim.y*blockDim.y; // Total amount of threads. 
    unsigned int offset{};
    unsigned int index = tidX*a.columns + tidY;    // we'll use cache memory L2 because of its rate of 2,000GBps higher thant GDRAM (300GBps) and PCIe (16GBps) - SM is at 20,000GBps
    
    extern __shared__ T cache[]; //size of ceil((ROWS*COLUMNS)/(gridX*gridY*blockX*blockY))
    
    #pragma unroll
    while (index+offset < a.rows*a.columns){
        cache[((threadIdx.x+blockDim.x*(offset/stride))*(blockDim.x-1)+ threadIdx.y)] = *(a+index+offset);
        offset += stride;
    }
    __syncthreads();
    offset = 0;
    #pragma unroll
    while (index+offset < a.rows*a.columns){
        *(b + tidY*amountRows + tidX + offset)  = cache[((threadIdx.x+blockDim.x*(offset/stride))*(blockDim.x-1)+ threadIdx.y)];
        offset += stride;
    }
}

__global__ void dot(const Matrix  __restrict__ a, const Matrix __restrict__  b, Matrix c) {
    /*
    Following the hypothesis that we are using digital type, double , float or int
    */
    unsigned int tidX = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int tidY = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned long int stride = gridDim.x*blockDim.x + gridDim.y*blockDim.y; // Total amount of threads. 
    unsigned int offset{};
    
    T intermediateValue{};
    if (tidX < a.columns && tidY < b.rows){
        #pragma unroll
        for (unsigned int i = 0; i < a.columns; i += 4){
            double4 a_tmp = reinterpret_cast<double4*>(&a.elements[i+a.columns*tidY])[0];
            double4 b_tmp = reinterpret_cast<double4*>(&b.elements[i+b.columns*tidY])[0];
            intermediateValue += (a_tmp.x * b_tmp.x);
            intermediateValue += (a_tmp.y * b_tmp.y);
            intermediateValue += (a_tmp.z  * b_tmp.z);
            intermediateValue += (a_tmp.w * b_tmp.w);
            }
        *c(tidY*a.rows + tidX = intermediateValue;
        }
}

int main(void){
    // Test
    return 0; 
}


#endif 